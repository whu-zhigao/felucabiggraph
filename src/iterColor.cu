#include "hip/hip_runtime.h"
#include<string.h>
#include<malloc.h>
#include<stdio.h>
#include<omp.h>
#include<hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <math.h>


#include "graph.h"
#include "timer.h"
#include "hip/hip_runtime.h"
#include "util.h"

// The number of partitioning the outer chunk must be greater or equal to 1
#define ITERATE_IN_OUTER 2
#define NUM_THREADS 1

#ifdef __CUDA_RUNTIME_H__
#define HANDLE_ERROR(err) if (err != hipSuccess) {	\
	printf("CUDA Error in %s at line %d: %s\n", \
			__FILE__, __LINE__, hipGetErrorString(hipGetLastError()));\
	exit(1);\
}
#endif  // #ifdef __CUDA_RUNTIME_H__  

static __global__ void  coloring_kernel_outer(  
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		const int * const out_degree,
		int * const values,
		int * const undone)
{
	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum=0.0f;

	//int delta = 0;
    //hiprandState localState;    
    //hiprand_init(clock64(),index,0,&localState);

	for (int i = index; i < edge_num; i+=n)
	{
		if(values[edge_src[i]] == values[edge_dest[i]])
		{
			/*
			//delta = hiprand(&localState);
			delta = hiprand(&localState) % 100;			
			//atomicAdd(&add_values[edge_dest[i]],delta);	//atomicAdd(&add_values[edge_dest[i]],delta) equals add_values[edge_dest[i]]+=delta;
			add_values[edge_dest[i]] = atomicAdd(&add_values[edge_dest[i]],delta) % 100;
			*/
			values[edge_dest[i]] = values[edge_src[i]] + 1;
			undone[edge_dest[i]] = 1;
		}

		//printf("vertex %d, values[edge_src]: %d, values[edge_dest]: %d \n", i, values[edge_src[i]], values[edge_dest[i]]);
	}
}

static __global__ void coloring_kernel_inner(  
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		const int * const out_degree,
		int * const values,
		int * const undone,
		int * continue_flag)
{
	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int flag=0;

	for (int i = index; i < edge_num; i+=n)
	{
		if(values[edge_src[i]] == values[edge_dest[i]])
		{
			values[edge_dest[i]] = values[edge_src[i]] + 1;
			undone[edge_dest[i]] = 1;
			*continue_flag = 1;
		}
	}
	__syncthreads();
	//check
	int new_value=0;
	for (int i = index; i < edge_num; i+=n)
	{	
		new_value = undone[edge_dest[i]];		

		if(new_value == values[edge_src[i]])
		{
			flag = 1;
		}
	}
	if (flag==1)  *continue_flag=1;
}

void merge_value_on_cpu(
		int const vertex_num, 
		int const gpu_num, 
		int * const  *h_undone, 
		int * const color_value_gpu , 
		int *copy_num, 
		int flag)
{
	int i,id;
	omp_set_num_threads(NUM_THREADS);	

	int temp_color;
    
#pragma omp parallel private(i)
	{
		id=omp_get_thread_num(); 
		for (i = id; i < vertex_num; i=i+NUM_THREADS)
		{
			if (copy_num[i]>1)
			{
				temp_color=h_undone[0][i];
				for (int j = 0; j < gpu_num; ++j)
				{
					if(temp_color < h_undone[j][i])
						temp_color = h_undone[j][i];
				}
				color_value_gpu[i] = temp_color;				
			}
			//colors[i] = color_value_gpu[i];
		}
		//printf("vertex_num is: %d, total color number is %d \n", vertex_num, countDistinct(colors, vertex_num));   
	}	
}

static __global__ void kernel_extract_values(
		int const edge_num,
		int * const edge_dest,
		int * const undone,
		int * const value
		)
{
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = index; i < edge_num; i+=n)
	{
		int dest=edge_dest[i];
		value[dest]=undone[dest];
		undone[dest]=0;
	}  
}

void Gather_result_color(
		int const vertex_num, 
		int const gpu_num, 
		int * const copy_num,
		int * const  *h_delta_undone,  
		int * const value_gpu
		)
{
	int i,id;
	int new_value=0;
	omp_set_num_threads(NUM_THREADS);	
#pragma omp parallel private(i)
	{
		id=omp_get_thread_num(); 
		for (i = id; i < vertex_num; i=i+NUM_THREADS)
		{
			if (copy_num[i]>1)
			{
				new_value=h_delta_undone[0][i];
				for (int j = 0; j < gpu_num; ++j)
				{
					if(new_value < h_delta_undone[j][i])
						new_value = h_delta_undone[j][i];					  
				}
				value_gpu[i]=new_value;	
			}
			//printf("Vertex ID: %d, Vertex Color: %d \n", i, value_gpu[i]); 
		}
	}
	//printf("vertex_num is: %d, total color number is %d \n", vertex_num, countDistinct(value_gpu, vertex_num));  
}

/* PageRank algorithm on GPU */
void coloring_gpu(Graph **g,int gpu_num,int *value_gpu,DataSize *dsize, int* out_degree, int *copy_num, int **position_id)
{
	printf("Graph Coloring is running on GPU...............\n");
	printf("Start malloc edgelist...\n");

	int **h_flag=(int **)malloc(sizeof(int *)*gpu_num);
	int vertex_num=dsize->vertex_num;
	int **d_edge_inner_src=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_inner_dst=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_outer_src=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_outer_dst=(int **)malloc(sizeof(int *)*gpu_num);
	int **h_value=(int **)malloc(sizeof(int *)* gpu_num);
	int **h_add_value=(int **)malloc(sizeof(int *)*gpu_num);

	int **d_value=(int **)malloc(sizeof(int *)*gpu_num);
	//pr different
	//int **d_tem_value=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_add_value=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_outdegree=(int **)malloc(sizeof(int *)*gpu_num);

	int **d_flag=(int **)malloc(sizeof(int *)*gpu_num);

	/* determine the size of outer vertex in one process*/
	int tmp_per_size = min_num_outer_edge(g,gpu_num);
	int outer_per_size=tmp_per_size/ITERATE_IN_OUTER;
	int iterate_in_outer=ITERATE_IN_OUTER+1;
	int *last_outer_per_size=(int *)malloc(sizeof(int)*gpu_num);
	memset(last_outer_per_size,0,sizeof(int)*gpu_num);

	for (int i = 0; i < gpu_num; ++i)
	{
		h_value[i]=(int *)malloc(sizeof(int)*(vertex_num+1));
		h_add_value[i]=(int *)malloc(sizeof(int)*(vertex_num+1));
		//memset 0.0 or 1.0 
		memset(h_value[i],0.0,sizeof(int)*(vertex_num+1));
		h_flag[i]=(int *)malloc(sizeof(int));
	}

	/*Cuda Malloc*/
	/* Malloc stream*/
	hipStream_t **stream;
	hipEvent_t tmp_start,tmp_stop;
	stream=(hipStream_t **)malloc(gpu_num*sizeof(hipStream_t*));

	hipEvent_t * start_outer,*stop_outer,*start_inner,*stop_inner,*start_asyn,*stop_asyn;
	start_outer=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_outer=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	start_inner=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_inner=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	start_asyn=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_asyn=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));

	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		stream[i]=(hipStream_t *)malloc((iterate_in_outer+1)*sizeof(hipStream_t));
		HANDLE_ERROR(hipEventCreate(&start_outer[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_outer[i],0));
		HANDLE_ERROR(hipEventCreate(&start_inner[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_inner[i],0));  
		HANDLE_ERROR(hipEventCreate(&start_asyn[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_asyn[i],0));


		for (int j = 0; j <= iterate_in_outer; ++j)
		{
			HANDLE_ERROR(hipStreamCreate(&stream[i][j]));
		}
	}

	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		int out_size=g[i]->edge_outer_num;
		int inner_size=g[i]->edge_num - out_size;

		HANDLE_ERROR(hipMalloc((void **)&d_edge_outer_src[i],sizeof(int)*out_size));
		HANDLE_ERROR(hipMalloc((void **)&d_edge_outer_dst[i],sizeof(int)*out_size));

		if (outer_per_size!=0 && outer_per_size < out_size)
		{
			for (int j = 1; j < iterate_in_outer; ++j)
			{
				HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_outer_src[i]+(j-1)*outer_per_size),(void *)(g[i]->edge_outer_src+(j-1)*outer_per_size),sizeof(int)*outer_per_size,hipMemcpyHostToDevice, stream[i][j-1]));
				HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_outer_dst[i]+(j-1)*outer_per_size),(void *)(g[i]->edge_outer_dst+(j-1)*outer_per_size),sizeof(int)*outer_per_size,hipMemcpyHostToDevice, stream[i][j-1]));			
			}
		}

		last_outer_per_size[i]=g[i]->edge_outer_num-outer_per_size * (iterate_in_outer-1);           
		if (last_outer_per_size[i]>0 && iterate_in_outer>1 )
		{
			HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_outer_src[i]+(iterate_in_outer-1)*outer_per_size),(void *)(g[i]->edge_outer_src+(iterate_in_outer-1)*outer_per_size),sizeof(int)*last_outer_per_size[i],hipMemcpyHostToDevice, stream[i][iterate_in_outer-1]));
			HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_outer_dst[i]+(iterate_in_outer-1)*outer_per_size),(void *)(g[i]->edge_outer_dst+(iterate_in_outer-1)*outer_per_size),sizeof(int)*last_outer_per_size[i],hipMemcpyHostToDevice, stream[i][iterate_in_outer-1]));
		}


		HANDLE_ERROR(hipMalloc((void **)&d_edge_inner_src[i],sizeof(int)*inner_size));
		HANDLE_ERROR(hipMalloc((void **)&d_edge_inner_dst[i],sizeof(int)*inner_size));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_edge_inner_src[i],(void *)g[i]->edge_inner_src,sizeof(int)*inner_size,hipMemcpyHostToDevice,stream[i][iterate_in_outer]));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_edge_inner_dst[i],(void *)g[i]->edge_inner_dst,sizeof(int)*inner_size,hipMemcpyHostToDevice,stream[i][iterate_in_outer]));

		HANDLE_ERROR(hipMalloc((void **)&d_value[i],sizeof(int)*(vertex_num+1)));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_value[i],(void *)h_value[i],sizeof(int)*(vertex_num+1),hipMemcpyHostToDevice,stream[i][0]));
		//pr different
		HANDLE_ERROR(hipMalloc((void **)&d_add_value[i],sizeof(int)*(vertex_num+1)));
		//"memset only works for bytes. If you're using the runtime API, you can use thrust::fill() instead"
		//HANDLE_ERROR(hipMemset((void **)&d_add_value[i],0,sizeof(int)*(vertex_num+1)));

		//HANDLE_ERROR(hipMalloc((void **)&d_tem_value[i],sizeof(int)*(vertex_num+1)));
		//HANDLE_ERROR(hipMalloc((void **)&d_tem_value[i],sizeof(int)*(vertex_num+1)));
		HANDLE_ERROR(hipMalloc((void **)&d_outdegree[i],sizeof(int)*(vertex_num+1)));
		HANDLE_ERROR(hipMemcpyAsync(d_outdegree[i],out_degree, sizeof(int)*(vertex_num+1),hipMemcpyHostToDevice,stream[i][0]));

		HANDLE_ERROR(hipMalloc((void **)&d_flag[i],sizeof(int)));


	}
	printf("Malloc is finished!\n");

	/* Before While: Time Initialization */
	float *outer_compute_time,*inner_compute_time,*compute_time,*total_compute_time,*extract_bitmap_time;
	float gather_time=0.0;
	float cpu_gather_time=0.0;
	float total_time=0.0;
	float record_time=0.0;
	outer_compute_time=(float *)malloc(sizeof(int)*gpu_num);
	inner_compute_time=(float *)malloc(sizeof(int)*gpu_num);
	compute_time=(float *)malloc(sizeof(int)*gpu_num);
	total_compute_time=(float *)malloc(sizeof(int)*gpu_num);
	extract_bitmap_time=(float *)malloc(sizeof(int)*gpu_num);

	memset(outer_compute_time,0,sizeof(int)*gpu_num);
	memset(inner_compute_time,0,sizeof(int)*gpu_num);
	memset(compute_time,0,sizeof(int)*gpu_num);


	/* Before While: Variable Initialization */
	int step=0;
	int flag=0;
	int inner_edge_num=0;

	printf("Computing......\n");
	do
	{
		flag=0;
		for (int i = 0; i <gpu_num; ++i)
		{		
			memset(h_flag[i],0,sizeof(int));
			hipSetDevice(i);
            HANDLE_ERROR(hipMemset(d_flag[i],0,sizeof(int)));
			HANDLE_ERROR(hipEventRecord(start_outer[i], stream[i][0]));
			//kernel of outer edgelist
			if (outer_per_size!=0 && outer_per_size < g[i]->edge_outer_num)
			{
				for (int j = 1; j < iterate_in_outer; ++j)
				{				
					coloring_kernel_outer<<<208,128,0,stream[i][j-1]>>>(
							outer_per_size,
							d_edge_outer_src[i]+(j-1)*outer_per_size,
							d_edge_outer_dst[i]+(j-1)*outer_per_size,
							d_outdegree[i],
							d_value[i],
							d_add_value[i]);
					//TODO didn't not realize overlap
					//HANDLE_ERROR(hipMemcpyAsync((void *)(h_add_value[i]),(void *)(d_add_value[i]),sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost,stream[i][j-1]));
				}
			}

			last_outer_per_size[i]=g[i]->edge_outer_num-outer_per_size * (iterate_in_outer-1);           
			if (last_outer_per_size[i]>0 && iterate_in_outer>1  )
			{
				coloring_kernel_outer<<<208,128,0,stream[i][iterate_in_outer-1]>>>(
						last_outer_per_size[i],
						d_edge_outer_src[i]+(iterate_in_outer-1)*outer_per_size,
						d_edge_outer_dst[i]+(iterate_in_outer-1)*outer_per_size,
						d_outdegree[i],
						d_value[i],
						d_add_value[i]);
				//TODO didn't not realize 
				//HANDLE_ERROR(hipMemcpyAsync((void *)(h_add_value[i]),(void *)(d_add_value[i]),sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost,stream[i][iterate_in_outer-1]));
			}
			HANDLE_ERROR(hipEventRecord(stop_outer[i], stream[i][iterate_in_outer-1]));

            HANDLE_ERROR(hipMemcpy((void *)(h_add_value[i]),(void *)(d_add_value[i]),sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipEventRecord(start_inner[i], stream[i][iterate_in_outer]));
			//inner+flag
			inner_edge_num=g[i]->edge_num-g[i]->edge_outer_num;
			if (inner_edge_num>0)
			{
				coloring_kernel_inner<<<208,128,0,stream[i][iterate_in_outer]>>>(
						inner_edge_num,
						d_edge_inner_src[i],
						d_edge_inner_dst[i],
						d_outdegree[i],
						d_value[i],
						d_add_value[i],
						d_flag[i]);			
				HANDLE_ERROR(hipMemcpyAsync(h_flag[i], d_flag[i],sizeof(int),hipMemcpyDeviceToHost,stream[i][iterate_in_outer]));	    
			}
			HANDLE_ERROR(hipEventRecord(stop_inner[i],stream[i][iterate_in_outer]));
		}


		//merge bitmap on gpu
		double t1=omp_get_wtime();
		merge_value_on_cpu(vertex_num, gpu_num, h_add_value, value_gpu, copy_num, flag);
		double t2=omp_get_wtime();
		record_time=(t2-t1)*1000;
		gather_time+=record_time;


		for (int i = 0; i < gpu_num; ++i)
		{
			hipSetDevice(i);
			//extract bitmap to the value
			HANDLE_ERROR(hipMemcpyAsync(d_add_value[i], value_gpu,sizeof(int)*(vertex_num+1),hipMemcpyHostToDevice,stream[i][0]));
			HANDLE_ERROR(hipEventRecord(start_asyn[i], stream[i][0]));
			// d_value copy to the value of outer vertices
			kernel_extract_values<<<208,128,0,stream[i][0]>>>
				(  
				 g[i]->edge_outer_num,
				 d_edge_outer_dst[i],
				 d_add_value[i],
				 d_value[i]
				);		
			HANDLE_ERROR(hipEventRecord(stop_asyn[i], stream[i][0]));
		}

		for (int i = 0; i < gpu_num; ++i)
		{
			flag=flag||h_flag[i][0];
		}
		step++;

		//collect time  different stream
		for (int i = 0; i < gpu_num; ++i)
		{
			hipSetDevice(i);
			HANDLE_ERROR(hipEventSynchronize(stop_outer[i]));
			HANDLE_ERROR(hipEventSynchronize(stop_inner[i]));
			HANDLE_ERROR(hipEventSynchronize(stop_asyn[i]));

			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_outer[i], stop_outer[i]));
			outer_compute_time[i]+=record_time;
			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_inner[i], stop_inner[i]));  
			inner_compute_time[i]+=record_time;
			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_asyn[i], stop_asyn[i]));  
			extract_bitmap_time[i]+=record_time;
			total_compute_time[i]=outer_compute_time[i]+extract_bitmap_time[i]-inner_compute_time[i]>0?(outer_compute_time[i]+extract_bitmap_time[i]):inner_compute_time[i];
		}		
	}while(flag && step<200);


	//Todo to get the true value of inner vertice and outer vertice
	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		hipMemcpyAsync((void *)h_value[i],(void *)d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost,stream[i][0]);
	}

	printf("Gather result on cpu....\n");
	Gather_result_color(vertex_num,gpu_num,copy_num,h_add_value,value_gpu);

	//printf("vertex_num is: %d, total color number is %d \n", vertex_num, countDistinct(value_gpu, vertex_num));

	printf("Time print\n");

	//collect the information of time 
	int total_time_n=0.0;
	for (int i = 0; i < gpu_num; ++i)
	{
		if(total_time_n<total_compute_time[i])
			total_time_n=total_compute_time[i];
	}
	total_time=total_time_n>gather_time?total_time_n:gather_time;

//	printf("Total time of coloring_gpu is %.3f ms\n",total_time);
	printf("Elapsed time of coloring is %.3f ms\n", total_time/(step));
	printf("-------------------------------------------------------\n");
	printf("Detail:\n");
	printf("\n");
	for (int i = 0; i < gpu_num; ++i)
	{
		printf("GPU %d\n",i);
		printf("Outer_Compute_Time(include pre-stage):  %.3f ms\n", outer_compute_time[i]/step);
		printf("Inner_Compute_Time:                     %.3f ms\n", inner_compute_time[i]/step);
		printf("Total Compute_Time                      %.3f ms\n", total_compute_time[i]/step);
		printf("Extract_Bitmap_Time                     %.3f ms\n", extract_bitmap_time[i]/step);
	}
	printf("CPU \n");
	printf("CPU_Gather_Time:                            %.3f ms\n", gather_time/step);
	printf("--------------------------------------------------------\n");

	//clean
	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		//HANDLE_ERROR(hipEventDestroy(start[i]));
		//HANDLE_ERROR(hipEventDestroy(stop[i]));
		HANDLE_ERROR(hipFree(d_edge_outer_src[i]));
		HANDLE_ERROR(hipFree(d_edge_outer_dst[i]));
		HANDLE_ERROR(hipFree(d_edge_inner_src[i]));
		HANDLE_ERROR(hipFree(d_edge_inner_dst[i]));
		HANDLE_ERROR(hipFree(d_value[i]));
		HANDLE_ERROR(hipFree(d_flag[i]));

		HANDLE_ERROR(hipDeviceReset());
		//error 
		//free(h_value[i]);
		free(h_flag[i]);
		free(stream[i]);
	}
	free(outer_compute_time);
	free(inner_compute_time);
	free(compute_time);
}
